
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

#define BLOCKSIZE 32

__global__ void _matmul(
    uint M,
    uint K,
    uint N,
    const float *A,
    const float *B,
    float *C
) {

    uint n = blockIdx.x * blockDim.x + threadIdx.x; // col
    uint m = blockIdx.y * blockDim.y + threadIdx.y; // row

    if (m < M && n < N) {

        float value = 0;

        for (int k = 0; k < K; ++k) {

            value += A[m * K + k] * B[k * N + n];
        }

        C[m * N + n] = value;
    }

    // const int cRow = blockIdx.x * BLOCKSIZE + (threadIdx.x / BLOCKSIZE);
    // const int cCol = blockIdx.y * BLOCKSIZE + (threadIdx.x % BLOCKSIZE);

    // // if statement is necessary to make things work under tile quantization
    // if (cRow < M && cCol < N) {

    //     float tmp = 0.0;

    //     for (int i = 0; i < K; ++i) {

    //         tmp += A[cRow * K + i] * B[i * N + cCol];
    //     }

    //     C[cRow * N + cCol] = tmp;
    // }
}

void matmul(
    uint M, // rows of C / rows of A
    uint K, // columns of A / rows of B
    uint N, // columns of C / columns of B
    const float *A,
    const float *B,
    float *C
) {

    hipEvent_t start;
    hipEvent_t stop;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    dim3 gridDim(ceil(M / BLOCKSIZE), ceil(M / BLOCKSIZE), 1);
    dim3 blockDim(BLOCKSIZE, BLOCKSIZE, 1);
    // dim3 blockDim(BLOCKSIZE * BLOCKSIZE, 1, 1);

    float* d_A;
    float* d_B;
    float* d_C;

    hipMalloc(&d_A, N * K * sizeof(float));
    hipMalloc(&d_B, K * M * sizeof(float));
    hipMalloc(&d_C, N * M * sizeof(float));

    hipMemcpy(d_A, A, N * K * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, K * M * sizeof(float), hipMemcpyHostToDevice);

    hipEventRecord(start);

    _matmul<<<gridDim, blockDim>>>(
        M,
        K,
        N,
        d_A,
        d_B,
        d_C
    );

    hipEventRecord(stop);

    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    std::cout << "custom kernel execution time: " << milliseconds << std::endl;

    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipMemcpy(C, d_C, N * M * sizeof(float), hipMemcpyDeviceToHost);
}
