
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void _matmul(
    uint M,
    uint K,
    uint N,
    const float *A,
    const float *B,
    float *C
) {

    uint n = blockIdx.x * blockDim.x + threadIdx.x; // col
    uint m = blockIdx.y * blockDim.y + threadIdx.y; // row

    float value = 0;

    for (uint k = 0; k < K; k++) {

        value += A[m * K + k] * B[k * N + n];
    }

    C[m * N + n] = value;
}

void matmul(
    uint M, // rows of C / rows of A
    uint K, // columns of A / rows of B
    uint N, // columns of C / columns of B
    const float *A,
    const float *B,
    float *C
) {

    hipEvent_t start;
    hipEvent_t stop;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    dim3 gridDim(M / 32, N / 32, 1);
    dim3 blockDim(32, 32, 1);

    float* d_A;
    float* d_B;
    float* d_C;

    hipMalloc(&d_A, N * K * sizeof(float));
    hipMalloc(&d_B, K * M * sizeof(float));
    hipMalloc(&d_C, N * M * sizeof(float));

    hipMemcpy(d_A, A, N * K * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, K * M * sizeof(float), hipMemcpyHostToDevice);

    hipEventRecord(start);

    _matmul<<<gridDim, blockDim>>>(
        M,
        K,
        N,
        d_A,
        d_B,
        d_C
    );

    hipEventRecord(stop);

    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    std::cout << "kernel execution time: " << milliseconds << std::endl;

    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipMemcpy(C, d_C, N * M * sizeof(float), hipMemcpyDeviceToHost);
}
