
#include <hip/hip_runtime.h>
#include <iostream>
#include <hipblas.h>

void matmul(
    uint M, // rows of C / rows of A
    uint K, // columns of A / rows of B
    uint N, // columns of C / columns of B
    const float *A,
    const float *B,
    float *C
) {

    hipEvent_t start;
    hipEvent_t stop;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipblasHandle_t handle;
    hipblasCreate(&handle);

    // dim3 gridDim(M / 32, N / 32, 1);
    // dim3 blockDim(32, 32, 1);

    float* d_A;
    float* d_B;
    float* d_C;

    hipMalloc(&d_A, N * K * sizeof(float));
    hipMalloc(&d_B, K * M * sizeof(float));
    hipMalloc(&d_C, N * M * sizeof(float));

    hipMemcpy(d_A, A, N * K * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, K * M * sizeof(float), hipMemcpyHostToDevice);

    float alpha = 1.0f;
    float beta = 0.0f;

    hipEventRecord(start);

    hipblasSgemm(
        handle,
        HIPBLAS_OP_N,
        HIPBLAS_OP_N,
        N,
        M,
        K,
        &alpha,
        d_B,
        N,
        d_A,
        K,
        &beta,
        d_C,
        N
    );

    hipEventRecord(stop);

    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    std::cout << "kernel execution time: " << milliseconds << std::endl;

    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipMemcpy(C, d_C, N * M * sizeof(float), hipMemcpyDeviceToHost);

    hipblasDestroy(handle);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}
